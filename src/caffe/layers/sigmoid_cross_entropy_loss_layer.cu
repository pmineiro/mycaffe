#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidCrossEntropyLossForward(const int n, const Dtype* in, const Dtype* target, Dtype* loss) {
  CUDA_KERNEL_LOOP(index, n) {
    // In[1] := Limit[(-p (1 - t) - Log[1 + Exp[-p]]) - (t p), p -> -Infinity] 
    // Out[1] = 0

    if (in[index] < -15) {
      loss[index] = in[index] * target[index];
    } else {
      loss[index] = -(-in[index] * (1. - target[index]) - log(1. + exp(-in[index])));
    }
  }
}

template <typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  Dtype* loss_data = loss_data_.mutable_gpu_data();
  SigmoidCrossEntropyLossForward<Dtype>
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, input_data, target, loss_data);
  CUDA_POST_KERNEL_CHECK;

  Dtype loss = 0;
  caffe_gpu_asum(count, loss_data, &loss);
  top[0]->mutable_cpu_data()[0] = loss / num;
}

template <typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const int num = bottom[0]->num();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);
    caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);
    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(count, loss_weight / num, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossLayer);


}  // namespace caffe
